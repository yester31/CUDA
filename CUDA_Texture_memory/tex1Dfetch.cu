#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <stdio.h>

#define N 1024
texture<float, 1, hipReadModeElementType> tex;

__global__ void kernel_tex1Dfetch()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float x = tex1Dfetch(tex, i);
}

int main() {
	float* buffer;
	hipMalloc(&buffer, N * sizeof(float));
	hipBindTexture(0, tex, buffer, N * sizeof(float));

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(N, 1, 1);
	kernel_tex1Dfetch << <dimGrid, dimBlock >> > ();
	hipUnbindTexture(tex);
	hipFree(buffer);



}

