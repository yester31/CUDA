#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <stdio.h>

#define W 256
#define H 256

texture<float, 2, hipReadModeElementType> tex;

__global__ void kernel()
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float v = tex2D (tex, x, y);
}

int main() {
	float* buffer;
	hipMalloc(&buffer, W * H * sizeof(float));
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	hipBindTexture2D(0, tex, buffer, desc, W, H, W * sizeof(float));

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(W, H, 1);
	kernel << <dimGrid, dimBlock >> > ();
	hipUnbindTexture(tex);
	hipFree(buffer);



}

