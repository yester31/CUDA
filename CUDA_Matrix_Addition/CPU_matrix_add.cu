
#include <hip/hip_runtime.h>
#include <cstdio>

void add(const int x, const int y,
	const int WIDTH, int*c, const int*a,const int*b) {
	int i = y * (WIDTH)+x;
	c[i] = a[i] + b[i];
}

int main(void) {

	//host-side data
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = {0};

	// make a, b matirces
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			a[y][x] = y * 10 + x;
			b[y][x] = (y * 10 + x) * 100;
		}
	}

	// calculate_method_1
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			c[y][x] = a[y][x] + b[y][x];
		}
	}

	// calculate_method_2
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			add(x, y, WIDTH, (int*)c, (int*)a, (int*)b);
		}
	}

	//print the result 
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			printf("%5d", c[y][x]);
		}printf("\n");
	}

	//done
	return 0;
}