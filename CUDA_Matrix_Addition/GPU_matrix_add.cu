#include "hip/hip_runtime.h"
// 2020-5-16 by YH 

#include "util_cuda.cuh"

//kernel program for the device (GPU): compiled by NVCC
__global__ void addKernel(int*c, const int* a, const int * b) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = y * (blockDim.x) + x; //[y][x] = y * WIDTH + x
	c[i] = a[i] + b[i];
}


int main(void) {
	//host-side data
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };

	// make a, b matirces
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			a[y][x] = y * 10 + x;
			b[y][x] = (y * 10 + x) * 100;
		}
	}

	//device-side data
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	// allocate device memory
	CUDA_CHECK(hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int)));

	//copy from host to device 
	CUDA_CHECK(hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));//dev_a=a;
	CUDA_CHECK(hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));//dev_b=b;

	//launch a kernel on the GPU with one thread for each element.
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(WIDTH, WIDTH, 1);//x,y,z
	addKernel << <dimGrid, dimBlock >> > (dev_c, dev_a, dev_b);
	CUDA_CHECK(hipPeekAtLastError());

	//copy from device to host
	CUDA_CHECK(hipMemcpy(c,  dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost));//c=dev_c;

	//free device memory
	CUDA_CHECK(hipFree(dev_c));
	CUDA_CHECK(hipFree(dev_a));
	CUDA_CHECK(hipFree(dev_b));

	//print the result 
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			printf("%5d", c[y][x]);
		}printf("\n");
	}

	return 0;
}