// 2020-5-13 by YH 

#include <cstdio>
#include "hip/hip_runtime.h"

int main()
{
    // host-side data
    const int SIZE = 5;
    const int a[SIZE] = { 1, 2, 3, 4, 5 }; //source data
    int b[SIZE] = { 0,0,0,0,0 }; //final destination  

    //print source
    printf("Before {%d,%d,%d,%d,%d}\n", b[0], b[1], b[2], b[3], b[4]);

    //device-side data
    int* dev_a = 0;
    int* dev_b = 0;

    // allocate device memory
    hipMalloc((void**)&dev_a, SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, SIZE * sizeof(int));

    //copy from host to device
    hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

    //copy from device to device
    hipMemcpy(dev_b, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToDevice);

    //copy from device to host
    hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyHostToHost);

    //free device memory
    hipFree(dev_a);
    hipFree(dev_b);

    //print source
    printf("After {%d,%d,%d,%d,%d}\n", b[0], b[1], b[2], b[3], b[4]);

    return 0;
}
