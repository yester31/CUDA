#include "hip/hip_runtime.h"
// 2020-5-14 by YH 

#include "hip/hip_runtime.h"
#include ""
#include <cstdio>

#if defined(NDEBUG)     //release mode
#define CUDA_CHECK(x) (x)   
#else                   // debug mode
//error check 
#define CUDA_CHECK(x)   do{\
    (x); \
    hipError_t e = hipGetLastError(); \
    if (e != hipSuccess) { \
        printf("cuda failure %s at %s:%d \n", \
        hipGetErrorString(e), \
            __FILE__, __LINE__); \
        exit(0); \
    } \
}while(0)
#endif

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x; // each thread knows its own index
	c[i] = a[i] + b[i];
}

int main(void) {

	const int size = 5;
	const int a[size] = { 1, 2, 3, 4, 5 };
	const int b[size] = { 10, 20, 30, 40, 50 };
	int c[size] = { 0 };

	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	CUDA_CHECK(hipMalloc((void**)&dev_a, size * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_b, size * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_c, size * sizeof(int)));
	CUDA_CHECK(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));

	addKernel << <1, size >> > (dev_c, dev_a, dev_b);
	CUDA_CHECK(hipPeekAtLastError());

	CUDA_CHECK(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dev_c));
	CUDA_CHECK(hipFree(dev_a));
	CUDA_CHECK(hipFree(dev_b));

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
}