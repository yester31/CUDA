#include "hip/hip_runtime.h"
// 2020-5-17 by YH 

#include "../util_cu/util_cuda.cuh"
 
//kernel program for the device (GPU): compiled by NVCC
__global__ void mulKernel(int*c, const int* a, const int * b, const int WIDTH) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0.0f;
	for (int k = 0; k < WIDTH; ++k) {
		float lhs = a[y * WIDTH + k];
		float rhs = b[k * WIDTH + x];
		sum += lhs * rhs;
	}
	c[y * WIDTH + x] = sum;
}


int main(void) {
	//host-side data
	const int WIDTH = 8;
	const int TILE_WIDTH = 4;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };

	// make a, b matirces
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			a[y][x] = y + x;
			b[y][x] = y + x;
		}
	}

	//device-side data
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	// allocate device memory
	CUDA_CHECK(hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int)));

	//copy from host to device 
	CUDA_CHECK(hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));//dev_a=a;
	CUDA_CHECK(hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));//dev_b=b;

	//launch a kernel on the GPU with one thread for each element.
	dim3 dimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);//x,y,z
	mulKernel << <dimGrid, dimBlock >> > (dev_c, dev_a, dev_b, WIDTH);
	CUDA_CHECK(hipPeekAtLastError());

	//copy from device to host
	CUDA_CHECK(hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost));//c=dev_c;

	//free device memory
	CUDA_CHECK(hipFree(dev_c));
	CUDA_CHECK(hipFree(dev_a));
	CUDA_CHECK(hipFree(dev_b));

	//print the result 
	for (int y = 0; y < WIDTH; ++y) {
		for (int x = 0; x < WIDTH; ++x) {
			printf("%5d", c[y][x]);
		}printf("\n");
	}

	return 0;
}